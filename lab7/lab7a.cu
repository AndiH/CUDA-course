#include "hip/hip_runtime.h"
#include <iostream>
#include "stdio.h"
#include <vector>
#include <hip/hip_runtime.h>
#include "TROOT.h"
#include "TApplication.h"
#include "TSystem.h"
#include "TMinuit.h"
#include "TRandom3.h"
#include "TVectorT.h"
#include "TCanvas.h"
#include "TH1D.h"
#include "math.h"
//#include "cuPrintf.cu"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

// helper function
TVectorD stdVectorToRootVector (std::vector<double> vector) {
	TVectorD tempVector(vector.size());
	for (unsigned int i = 0; i < vector.size(); i++) tempVector[i] = vector[i];
	return tempVector;
}


std::vector<double> theEvents;

__constant__ __device__ double dev_params[5];
thrust::device_vector<double>* d_theEvents;

__device__ double dev_gaussian (double x, double mean, double sigma) {
	return exp(-0.5*pow((x - mean)/sigma, 2)) / (sigma * sqrt(2 * M_PI));
}

struct GaussianFunctor {
	__device__ double operator() (double x) {
		double mean1 = dev_params[0];
		double sigma1 = dev_params[1];
		double mean2 = dev_params[2];
		double sigma2 = dev_params[3];
		double weight1 = dev_params[4];
		
		return -2 * log(weight1 * dev_gaussian(x, mean1, sigma1)
				+ (1 - weight1) * dev_gaussian(x, mean2, sigma2)
			       );
	}
};

void dev_FitFcn (int& npar, double* deriv, double& fun, double* param, int flg) {
	hipMemcpyToSymbol(HIP_SYMBOL("dev_params"), param, 5*sizeof(double), 0, hipMemcpyHostToDevice);
	fun = thrust::transform_reduce(d_theEvents->begin(), d_theEvents->end(), GaussianFunctor(), 0., thrust::plus<double>());
}

int main(int argc, char** argv) {
// 	gSystem->Load("libMinuit");
	std::cout << "############################" << std::endl << "## You're lucky! Because of the default TMinuit output into the shell, I implemented a bunch of line separators!" << std::endl << "############################" << std::endl << std::endl;
	int sizeOfVector = 10000;
	if (argc > 1) sizeOfVector = atoi(argv[1]);
	
	TRandom3 myRandom(23);
	double myMean1 = 3;
	double mySigma1 = 0.6;
	double myMean2 = 2;
	double mySigma2 = 0.2;
	double myG1DrawProbability = 0.42; // (0,1]
	std::cout << "Mean1 = " << myMean1 << ", mySigma1 = " << mySigma1 << ", myMean2 = " << myMean2 << ", myMean2 = " << myMean2 << ",  weight1 = " << myG1DrawProbability << std::endl;
	for (int i = 0; i < sizeOfVector; i++) {
		if (myRandom.Uniform() <= myG1DrawProbability) {
			theEvents.push_back(myRandom.Gaus(myMean1, mySigma1));
		} else {
			theEvents.push_back(myRandom.Gaus(myMean2, mySigma2));
		}
	}
	
	thrust::device_vector<double> d_localEvents(theEvents);
	d_theEvents = &d_localEvents;
	
	TMinuit minuit(5);
	std::cout << "## TMINUIT:: Defining parameters ##" << std::endl;
	// DefineParameter syntax is: 
	// 	int paramter number, 
	//	char parmeter name,
	//	double initial value,
	//	double initial error,
	//	double lower limit,
	//	double upper limit
	minuit.DefineParameter(0, "mean1", myMean1, 0.1, myMean1-1, myMean2+1); // add +-2 for uncertainties
	minuit.DefineParameter(1, "sigma1", mySigma1, 0.1, mySigma1-1, mySigma1+1);
	minuit.DefineParameter(2, "mean2", myMean2, 0.1, myMean2-1, myMean2+1);
	minuit.DefineParameter(3, "sigma3", mySigma2, 0.1, mySigma2-1, mySigma2+1);
	minuit.DefineParameter(4, "weight1", 0.5, 0.01, 0., 1.);
	
	
	std::cout << "## TMINUIT:: Setting Function ##" << std::endl;
	minuit.SetFCN(&dev_FitFcn);
	std::cout << "## TMINUIT:: Calling Migrad() ##" << std::endl;
	minuit.Migrad();
	
// 	TVectorD root_theEvents = stdVectorToRootVector(theEvents);
	TH1D * histVis = new TH1D(stdVectorToRootVector(theEvents));
	TH1D * secondHist = new TH1D("secondHist", "Titel", 100, 1, 4);
	for (int i = 0; i < theEvents.size(); i++) secondHist->Fill(theEvents[i]);
	
	TApplication *theApp = new TApplication("app", &argc, argv, 0, -1);
	TCanvas * c1 = new TCanvas("c1", "default", 100, 10, 800, 600);
	secondHist->Draw("hist");
	c1->Update();
// 	c1->Print("c1.pdf");
	theApp->Run();

	
}