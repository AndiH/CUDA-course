#include "hip/hip_runtime.h"
#include <iostream>
#include "stdio.h"
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <hip/hip_runtime.h>
#include <cassert>
// #include "/private/herten/NVIDIA_GPU_Computing_SDK/C/src/simplePrintf/cuPrintf.cuh"
#include "../cuPrintf.cuh"


__global__ void dotProduct (int* vec1, int* vec2, int* vec3, int streamId) {
	vec3[threadIdx.x] = vec1[threadIdx.x] * vec2[threadIdx.x];
	printf("[%i]-[%i] vec1 * vec2 = vec3 == %i * %i = %i\n", streamId, threadIdx.x, vec1[threadIdx.x], vec2[threadIdx.x], vec3[threadIdx.x]);
}


int main (int argc, char** argv) {
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	assert(properties.deviceOverlap);
	
	hipStream_t stream0;
	hipStream_t stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	
	int sizeOfVector = 100;
	int nOfChunks = 4;
	if (argc > 1) nOfChunks = atoi(argv[1]);
	int chunkSize = sizeOfVector/nOfChunks;
	std::cout << "Using " << nOfChunks << " chunks, leading to a chunk size of " << chunkSize << "." << std::endl;
	
	int *host_a, *host_b, *host_result;
	int *dev0_a, *dev0_b, *dev0_result;
	int *dev1_a, *dev1_b, *dev1_result;
	
	hipHostAlloc((void**) &host_a, sizeOfVector*sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**) &host_b, sizeOfVector*sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**) &host_result, sizeOfVector*sizeof(int), hipHostMallocDefault);
	
	hipMalloc((void**) &dev0_a, chunkSize*sizeof(int));
	hipMalloc((void**) &dev0_b, chunkSize*sizeof(int));
	hipMalloc((void**) &dev0_result, chunkSize*sizeof(int));
	
	hipMalloc((void**) &dev1_a, chunkSize*sizeof(int));
	hipMalloc((void**) &dev1_b, chunkSize*sizeof(int));
	hipMalloc((void**) &dev1_result, chunkSize*sizeof(int));
	
	srand(23);
	for (unsigned int i = 0; i < sizeOfVector; i++) {
		host_a[i] = rand() % 100;
		host_b[i] = rand() % 100;
	}
	for (unsigned int i = 0; i < sizeOfVector; i++) {
		std::cout << "host_a[" << i << "] = " << host_a[i] << ", host_b[" << i << "] = " << host_b[i] << std::endl;
	}
	
	int numBlocks = 1;
	int numThreads = chunkSize;
	
	for (int i = 0; i < sizeOfVector; i += 2*chunkSize) {
		hipMemcpyAsync(dev0_a, host_a + i,
				chunkSize*sizeof(int),
				hipMemcpyHostToDevice,
				stream0);
		hipMemcpyAsync(dev1_a, host_a + chunkSize + i,
				chunkSize*sizeof(int),
				hipMemcpyHostToDevice,
				stream1);
		
		hipMemcpyAsync(dev0_b, host_b + i,
				chunkSize*sizeof(int),
				hipMemcpyHostToDevice,
				stream0);
		hipMemcpyAsync(dev1_b, host_b + chunkSize + i,
				chunkSize*sizeof(int),
				hipMemcpyHostToDevice,
				stream1);
		
		dotProduct<<<numBlocks, numThreads, 0, stream0>>>(dev0_a, dev0_b, dev0_result, i);
		dotProduct<<<numBlocks, numThreads, 0, stream1>>>(dev1_a, dev1_b, dev1_result, i+1);
		
		hipMemcpyAsync(host_result + i, dev0_result, chunkSize*sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_result + chunkSize + i, dev1_result, chunkSize*sizeof(int), hipMemcpyDeviceToHost, stream1);
	}
	
	hipDeviceSynchronize(); // wait for all threads to finish
	
	for (int i = 0; i < sizeOfVector; i++) {
		std::cout << "host[" << i << "] = " << host_result[i] << std::endl;
	}
	
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipFree(dev0_a);
	hipFree(dev0_b);
	hipFree(dev0_result);
	hipFree(dev1_a);
	hipFree(dev1_b);
	hipFree(dev1_result);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);
}
